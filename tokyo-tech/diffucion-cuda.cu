#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define NX 8192
#define NY 8192

float data[2][NY][NX];

/* in microseconds (us) */
double get_elapsed_time(struct timeval *begin, struct timeval *end)
{
    return (end->tv_sec - begin->tv_sec) * 1000000
            + (end->tv_usec - begin->tv_usec);
}

void init()
{
  int x, y;
  int cx = NX/2, cy = 0; /* center of ink */
  int rad = (NX+NY)/8; /* radius of ink */

  for(y = 0; y < NY; y++) {
    for(x = 0; x < NX; x++) {
      float v = 0.0;
      if (((x-cx)*(x-cx)+(y-cy)*(y-cy)) < rad*rad) {
	v = 1.0;
      }
      data[0][y][x] = v;
      data[1][y][x] = v;
    }
  }
  return;
}

__global__ void calcXY(int from, to) {
    //yのループを呼び出す
    int startx = threadIdx.x * 32 - 1;
    int y = blockIdx.y;
    int x;
    for (x = startx; x < startx + 32; x++) {
        if (x == 8191 || y == 8191 || x == 0 || y == 0) {
            continue;
        }
        data[to][y][x] = 0.2 * (data[from][y][x]
                    + data[from][y][x-1]
                    + data[from][y][x+1]
                    + data[from][y-1][x]
                    + data[from][y+1][x]);
    }
}

/* Calculate for one time step */
/* Input: data[t%2], Output: data[(t+1)%2] */
void calc(int nt)
{
  int t;
//dataのコピーをgpuに作成
  hipMalloc(data);
  for (t = 0; t < nt; t++) {
    int from = t%2;
    int to = (t+1)%2;

#if 1
    printf("step %d\n", t);
    //fflush(0);
#endif
    //並列でcalcXYを呼ぶ
    //1スレッドに対して、y一列計算する→xがいらなくなる
    calcXY<<<8192, 32<<<(from, to);
  }

  return;
}

int  main(int argc, char *argv[])
{
  struct timeval t1, t2;
  int nt = 20; /* number of time steps */
  
  if (argc >= 2) { /* if an argument is specified */
      nt = atoi(argv[1]);
  }

  init();

  gettimeofday(&t1, NULL);

  calc(nt);

  gettimeofday(&t2, NULL);

  {
      double us;
      double gflops;
      int op_per_point = 5; // 4 add & 1 multiply per point

      us = get_elapsed_time(&t1, &t2);
      printf("Elapsed time: %.3lf sec\n", us/1000000.0);
      gflops = ((double)NX*NY*nt*op_per_point)/us/1000.0;
      printf("Speed: %.3lf GFlops\n", gflops);
  }

  return 0;
}
